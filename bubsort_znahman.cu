
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <ctime>
#include <cstdlib>

/*
 * Class: CSCI563 - Introduction to Parallel Computing
 * Student: Zachary Nahman
 * Professor: Dr. Wu
 * Assignment: Course Project for Graduate Students
 * Due Date: 5/6/2018
*/

/*
 * TODO:
 * - take a positive integer N as an argument
 * - create an input integer array of size N
 * - populate the array with integers from the range [1,1000]
 * - sort the array using sequential bubblesort
 * - sort the array using parallel bubblesort
 * - compare sequential bubblesort and paralle bubblesort
*/


// CUDA kernel
__global__ void even_swapper(int *X, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i % 2 == 0 && i < N-1){
        if(X[i+1] < X[i]){
            // switch in the x array
            int temp = X[i];
            X[i] = X[i+1];
            X[i+1] = temp;
        }
    }
}

__global__ void odd_swapper(int *X, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i % 2 != 0 && i < N-2){
        if(X[i+1] < X[i]){
            // switch in the x array
            int temp = X[i];
            X[i] = X[i+1];
            X[i+1] = temp;
        }
    }
}


int main( int argc, char* argv[] )
{
    int N;
    // get the command line argument N (the size of the array)
    if(argc == 2){
        N = atoi(argv[1]);
    }
    else if(argc == 1){
        std::cout << "No number entered for N - please run with N specified" << "\n";
    }
    else if(argc > 2){
        std::cout << "Too many arguments entered, expected 1 - the array size" << "\n";
    }

    // declare the host input array
    int *h_input_array;
    size_t bytes = N * sizeof(int);
    h_input_array= (int*)malloc(bytes);

    // declare the host output array
    int *h_output_array;
    int *seq_h_output_array;
    h_output_array = (int*)malloc(bytes);
    seq_h_output_array = (int*)malloc(bytes);

    // fill the host input array with integers between [1,1000]
    // seed random number generator
    srand(time(0));

    //std::cout << "Array: " << "\n";
    for(int i = 0; i < N; i++){
         h_input_array[i] = (rand() % 1000) + 1;
         //std::cout << h_input_array[i] << "\n";
    }

    // sequential bubblesort algorithm
    // copy the array to sort
    for(int k = 0; k < N; k++){
         seq_h_output_array[k] = h_input_array[k];
         //std::cout << seq_h_output_array[k] << "\n";
    }
    // sort seq_h_output_array (with bubble sort)
    clock_t seq_begin = clock();
    bool sorted = false;
    while(!sorted){
        bool swapped = false;
        for(int k = 0; k < N-1; k++){
             if(seq_h_output_array[k+1] < seq_h_output_array[k]){
                 int temp = seq_h_output_array[k];
                 seq_h_output_array[k] = seq_h_output_array[k+1];
                 seq_h_output_array[k+1] = temp;
                 swapped = true;
             }
        }
        if(!swapped){
            sorted = true;
        }
    }
    clock_t seq_end = clock();


    // print results of sequential bubble sort for debugging
    // std::cout << "Sequential Array (sorted with bubblesort):" << "\n";
    // for(int k = 0; k < N; k++){
    //      std::cout << seq_h_output_array[k] << "\n";
    // }

    // declare and allocate device memory for arrays
    int *d_input_array;
    int *d_output_array;

    hipMalloc(&d_input_array, bytes);
    hipMalloc(&d_output_array, bytes);

    // Copy host input array to device
    hipMemcpy(d_input_array, h_input_array, bytes, hipMemcpyHostToDevice);

    int threadsToLaunch = ceil(N/32.0);
    //invoke the kernel function
    clock_t par_begin = clock();
    for(int i = 0; i < N;  i++){
        even_swapper<<<threadsToLaunch, 32>>>(d_input_array, N);
        odd_swapper<<<threadsToLaunch, 32>>>(d_input_array, N);
    }
    clock_t par_end = clock();

    // Copy array back to host
    hipMemcpy(h_output_array, d_input_array, bytes, hipMemcpyDeviceToHost);

    // printing parallel result for debugging
    // std::cout << "Parallel: " << "\n";
    // for(int k = 0; k < N; k++){
    //      std::cout << h_output_array[k] << "\n";
    // }

    // confirm both arrays are sorted:
    bool seq_sorted = true;
    bool par_sorted = true;
    for(int i = 0; i < N-1;  i++){
        if(h_output_array[i] > h_output_array[i+1]){
            par_sorted = false;
        }
        if(seq_h_output_array[i] > seq_h_output_array[i+1]){
            seq_sorted = false;
        }
    }

    if(seq_sorted){
        std::cout << "The sequential array is sorted properly!" << "\n";
    }else{
        std::cout << "The sequential array is NOT sorted properly!" << "\n";
    }

    if(par_sorted){
        std::cout << "The parallel array is sorted properly!" << "\n";
    }else{
        std::cout << "The parallel array is NOT sorted properly!" << "\n";
    }


    double seq_elapsed_secs = double(seq_end - seq_begin)/CLOCKS_PER_SEC;
    std::cout << "\n";
    std::cout << "Elapsed Time for Sequential Bubblesort: ";
    std::cout << seq_elapsed_secs;
    std::cout << " seconds";
    std::cout << "\n";

    double par_elapsed_secs = double(par_end - par_begin)/CLOCKS_PER_SEC;
    std::cout << "\n";
    std::cout << "Elapsed Time for Parallel Bubblesort: ";
    std::cout << par_elapsed_secs;
    std::cout << " seconds";
    std::cout << "\n";

    // Release device memory
    hipFree(d_input_array);
    hipFree(d_output_array);

    // Release host memory
    free(h_input_array);
    free(h_output_array);
    free(seq_h_output_array);

    return 0;
}
